#include "hip/hip_runtime.h"
/* This code accompanies
*   Two relaxation time lattice Boltzmann method coupled to fast Fourier transform Poisson solver: Application to electroconvective flow, Journal of Computational Physics
*	 https://doi.org/10.1016/j.jcp.2019.07.029
*	 Numerical analysis of electroconvection in cross-flow with unipolar charge injection, Physical Review Fluids
*	 https://doi.org/10.1103/PhysRevFluids.4.103701
*
*   Yifei Guan, Igor Novosselov
* 	 University of Washington
*
* Author: Yifei Guan
*
*/
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <stdlib.h>

#include "seconds.h"
#include "LBM.h"
#include "LBM.cu"
#include "poisson.cu"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

int main(int argc, char* argv[])
{
	hipMemcpyFromSymbol(&dt_host, HIP_SYMBOL(dt), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Lx_host, HIP_SYMBOL(Lx), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&Ly_host, HIP_SYMBOL(Ly), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&dy_host, HIP_SYMBOL(dy), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&voltage_host, HIP_SYMBOL(voltage), sizeof(double), 0, hipMemcpyDeviceToHost);
	hipMemcpyFromSymbol(&voltage2_host, HIP_SYMBOL(voltage2), sizeof(double), 0, hipMemcpyDeviceToHost);


	hipMemcpyToSymbol(HIP_SYMBOL(nu), &nu_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(uw), &uw_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(exf), &exf_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(K), &K_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(D), &D_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(Kn), &Kn_host, sizeof(double), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(epsn), &epsn_host, sizeof(double), 0, hipMemcpyHostToDevice);






	// Compute parameters
	compute_parameters(T, M, C, Fe);

    printf("Simulating Electro-Thermo-convection in 2D\n");
    printf("      domain size: %ux%u\n",NX,NY);
    printf("                T: %g\n",*T);
    printf("                M: %g\n",*M);
    printf("                C: %g\n",*C);
    printf("               Fe: %g\n",*Fe);
	printf("               Ra: %g\n", *Ra);
	printf("               Pr: %g\n", *Pr);
    printf("        timesteps: %u\n",NSTEPS);
    printf("       save every: %u\n",NSAVE);
    printf("    message every: %u\n",NMSG);
    printf("\n");
    
    double bytesPerMiB = 1024.0*1024.0;
    double bytesPerGiB = 1024.0*1024.0*1024.0;
    
    checkCudaErrors(hipSetDevice(0));
    int deviceId = 0;
    checkCudaErrors(hipGetDevice(&deviceId));
    
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));
    
    size_t gpu_free_mem, gpu_total_mem;
    checkCudaErrors(hipMemGetInfo(&gpu_free_mem,&gpu_total_mem));
    
    printf("CUDA information\n");
    printf("       using device: %d\n", deviceId);
    printf("               name: %s\n",deviceProp.name);
    printf("    multiprocessors: %d\n",deviceProp.multiProcessorCount);
    printf(" compute capability: %d.%d\n",deviceProp.major,deviceProp.minor);
    printf("      global memory: %.1f MiB\n",deviceProp.totalGlobalMem/bytesPerMiB);
    printf("        free memory: %.1f MiB\n",gpu_free_mem/bytesPerMiB);
    printf("\n");

	// storage of f0 at upper and lower plate
	checkCudaErrors(hipMalloc((void**)&f0bc, sizeof(double)*NX * 2));

    //double *prop_gpu;
	// microscopic variables
	checkCudaErrors(hipMalloc((void**)&f0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&f1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&f2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&h0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&h1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&h2_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&temp0_gpu, mem_size_0dir));
	checkCudaErrors(hipMalloc((void**)&temp1_gpu, mem_size_n0dir));
	checkCudaErrors(hipMalloc((void**)&temp2_gpu, mem_size_n0dir));
	// macroscopic variables
	checkCudaErrors(hipMalloc((void**)&rho_gpu,    mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uy_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&charge_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&phi_gpu,    mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&T_gpu,		mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&Ex_gpu,     mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&Ey_gpu,     mem_size_scalar));
	// Setup the cuFFT plan
	CHECK_CUFFT(hipfftPlan2d(&plan, NE, NX, HIPFFT_Z2Z));
	checkCudaErrors(hipMalloc((void**)&kx, sizeof(double)*NX));
	checkCudaErrors(hipMalloc((void**)&ky, sizeof(double)*NE));
	double *kx_host = (double*)malloc(sizeof(double)*NX);
	double *ky_host = (double*)malloc(sizeof(double)*NE);

	// Setup the frequencies kx and ky
	for (unsigned i = 0; i <= NX / 2; i++)
	{
		kx_host[i] = (double)i * 2.0 * M_PI / Lx_host;
	}

	for (unsigned i = NX / 2 + 1; i < NX; i++)
	{
		kx_host[i] = ((double)i - NX) * 2.0 * M_PI / Lx_host;
	}
	for (unsigned i = 0; i <= NE / 2; i++)
	{
		ky_host[i] = (double)i  * 2.0 * M_PI / (NE*dy_host);
	}

	for (unsigned i = NE / 2 + 1; i < NE; i++)
	{
		ky_host[i] = ((double)i - NE) * 2.0 * M_PI / (NE*dy_host);
	}

	CHECK(hipMemcpy(kx, kx_host,
		sizeof(double) * NX, hipMemcpyHostToDevice));
	CHECK(hipMemcpy(ky, ky_host,
		sizeof(double) * NE, hipMemcpyHostToDevice));
	
    //const size_t mem_size_props = 7*NX/nThreads*NY*sizeof(double);
    //checkCudaErrors(hipMalloc((void**)&prop_gpu,mem_size_props));
    
    double *scalar_host  = (double*) malloc(mem_size_scalar);
    if(scalar_host == NULL)
    {
        fprintf(stderr,"Error: unable to allocate required host memory (%.1f MiB).\n",mem_size_scalar/bytesPerMiB);
        exit(-1);
    }



	size_t total_mem_bytes = 2*mem_size_0dir + 4 * mem_size_n0dir + 7 * mem_size_scalar;// +mem_size_props;

    // create event objects
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));

	// Zero flow at t=0
	// to initialize rho, charge, phi, ux, uy, Ex, Ey fields.
	if (flag == 1) {
		read_data(&t, rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu, T_gpu);
	}
	else {
		initialization(rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu, T_gpu, kx, ky, plan);
		t = 0;
	}

	// initialise f1,h1 as equilibrium for rho, ux, uy, charge, ex, ey
    init_equilibrium(f0_gpu,f1_gpu,h0_gpu,h1_gpu, temp0_gpu, temp1_gpu, rho_gpu,charge_gpu, ux_gpu,uy_gpu,Ex_gpu,Ey_gpu, T_gpu);
    
	// open file for writing
	FILE *fout = fopen("data.dat", "wb+");
	save_data_tecplot(fout, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu, T_gpu, 1);
	// file for saving
	FILE *fumax = fopen("umax.dat", "wb+");
    
	// report computational results to screen
    report_flow_properties(0, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu,uy_gpu, Ex_gpu, Ey_gpu);
    
    double begin = seconds();
    checkCudaErrors(hipEventRecord(start,0));


    // main simulation loop; take NSTEPS time steps
    for(unsigned int i = 0; i <= NSTEPS; ++i)
    {
        // stream and collide from f1 storing to f2
        // optionally compute and save moments
        stream_collide_save(f0_gpu,f1_gpu,f2_gpu, h0_gpu, h1_gpu, h2_gpu, temp0_gpu, temp1_gpu, temp2_gpu, rho_gpu,charge_gpu,
			ux_gpu,uy_gpu, Ex_gpu, Ey_gpu, T_gpu, t, f0bc);
		// =========================================================================
		// Fast poisson solver
		// =========================================================================
		fast_Poisson(charge_gpu, T_gpu, kx, ky, plan);

		// =========================================================================
		// Save data for analysis
		// =========================================================================
       
        if(i%NSAVE == 1)
        {
			save_data_tecplot(fout, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu,T_gpu, 1);
		}

		if (i%NDMD == 1)
		{
			printf("Iteration: %u, physical time: %g.\n", i, t);
			// save for MATLAB postprocessing
			char filename[128];
			//sprintf(filename, "%g.dat", t);
			sprintf(filename, "charge_data");
			FILE *fout2 = fopen(filename, "ab");
			save_data_dmd(fout2, t, ux_gpu, uy_gpu, charge_gpu, phi_gpu);
			fclose(fout2);
		}
		if (i%printCurrent == 1) {
			checkCudaErrors(hipMemcpy(charge_host, charge_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
			checkCudaErrors(hipMemcpy(Ey_host, Ey_gpu, mem_size_scalar, hipMemcpyDeviceToHost));
			double current_host = current(charge_host, Ey_host);
			printf("Iteration: %u, physical time: %g, Current = %g\n", i, t, current_host);
			//printf("%g\n", Ez_host[scalar_index(0, 0, 0)]);
			// =============================================================================================================
			// save umax
			// =============================================================================================================
			record_umax(fumax, t, ux_gpu, uy_gpu);
		}        
		t = t + dt_host;
    }
    checkCudaErrors(hipEventRecord(stop,0));
    checkCudaErrors(hipEventSynchronize(stop));
    float milliseconds = 0.0f;
    checkCudaErrors(hipEventElapsedTime(&milliseconds,start,stop));

    double end = seconds();
    double runtime = end-begin;
    double gpu_runtime = 0.001*milliseconds;

    size_t doubles_read = ndir; // per node every time step
    size_t doubles_written = ndir;
    size_t doubles_saved = 3; // per node every NSAVE time steps
    
    // note NX*NY overflows when NX=NY=65536
    size_t nodes_updated = NSTEPS*size_t(NX*NY);
    size_t nodes_saved   = (NSTEPS/NSAVE)*size_t(NX*NY);
    double speed = nodes_updated/(1e6*runtime);

    double bandwidth = (nodes_updated*(doubles_read + doubles_written)+nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);

    printf(" ----- performance information -----\n");
    printf("  memory allocated (GPU): %.1f (MiB)\n",total_mem_bytes/bytesPerMiB);
    printf(" memory allocated (host): %.1f (MiB)\n",mem_size_scalar/bytesPerMiB);
    printf("               timesteps: %u\n",NSTEPS);
    printf("           clock runtime: %.3f (s)\n",runtime);
    printf("             gpu runtime: %.3f (s)\n",gpu_runtime);
    printf("                   speed: %.2f (Mlups)\n",speed);
    printf("               bandwidth: %.1f (GiB/s)\n",bandwidth);
    
	save_data_tecplot(fout, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu, T_gpu, 0);
	fclose(fout);
	// =============================================================================================================
	// save umax
	// =============================================================================================================
	record_umax(fumax, t, ux_gpu, uy_gpu);
	fclose(fumax);


	FILE *fend = fopen("data_end.dat", "wb+");

	save_data_end(fend, t, rho_gpu, charge_gpu, phi_gpu, ux_gpu, uy_gpu, Ex_gpu, Ey_gpu, T_gpu);
	fclose(fend);

    // destory event objects
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));
    
    // free all memory allocatd on the GPU and host
    checkCudaErrors(hipFree(f0_gpu));
    checkCudaErrors(hipFree(f1_gpu));
    checkCudaErrors(hipFree(f2_gpu));
	checkCudaErrors(hipFree(h0_gpu));
	checkCudaErrors(hipFree(h1_gpu));
	checkCudaErrors(hipFree(h2_gpu));
    checkCudaErrors(hipFree(rho_gpu));
	checkCudaErrors(hipFree(phi_gpu));
	checkCudaErrors(hipFree(Ex_gpu));
	checkCudaErrors(hipFree(Ey_gpu));
    checkCudaErrors(hipFree(ux_gpu));
    checkCudaErrors(hipFree(uy_gpu));
	checkCudaErrors(hipFree(f0bc));
	checkCudaErrors(hipFree(kx));
	checkCudaErrors(hipFree(ky));
	CHECK_CUFFT(hipfftDestroy(plan));
    //checkCudaErrors(hipFree(prop_gpu));    
    free(scalar_host);
	free(kx_host);
	free(ky_host);
    
    // release resources associated with the GPU device
    hipDeviceReset();
	system("pause");
    return 0;
}

